#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>

#define BOUND     2.0
#define EPS       1e-9

#define ITERS     16384
#define MAS_DIM   8192
#define BLOCK_DIM 16

#define complexEq(rel, iml, rer, imr) \
    abs(rel - rer) < EPS && abs(iml - imr) < EPS

__global__ void
getRoot(unsigned char * o_data)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    double cur_re = (double)(x - MAS_DIM / 2) / (MAS_DIM / (BOUND * 2));
    double cur_im = (double)(y - MAS_DIM / 2) / (MAS_DIM / (BOUND * 2));

    double new_re;
    double new_im;

    double num_re;
    double num_im;
    
    double div_re;
    double div_im;

    double div;

    for (unsigned int i = 0; i < ITERS; ++i)
    {
        // new_z = cur_z - (cur_z^3 - 1) / (3 * cur_z^2)
        div = 3.0 * (cur_re * cur_re * cur_re * cur_re +
                    2.0 * cur_re * cur_re * cur_im * cur_im +
                    cur_im * cur_im * cur_im * cur_im);
        
        if (complexEq(div, 0.0, 0.0, 0.0))
        {
            o_data[y * MAS_DIM + x] = 0 + '0';
            return;
        }

        div_re = cur_re * cur_re - cur_im * cur_im;
        div_im = 2.0 * cur_re * cur_im;

        num_re = cur_re * cur_re * cur_re - 3.0 * cur_re * cur_im * cur_im - 1.0;
        num_im = 3.0 * cur_re * cur_re * cur_im - cur_im * cur_im * cur_im;
        
        new_re = cur_re - (num_re * div_re + num_im * div_im) / div;
        new_im = cur_im - (div_re * num_im - div_im * num_re) / div;

        if (complexEq(new_re, new_im, cur_re, cur_im))
        {
            if (complexEq(new_re, new_im, 1.0, 0.0))
            {
                o_data[y * MAS_DIM + x] = 1 + '0';
            }
            else if (complexEq(new_re, new_im, -0.5, sqrt(3.0) / 2.0))
            {
                o_data[y * MAS_DIM + x] = 2 + '0';
            }
            else if (complexEq(new_re, new_im, -0.5, -sqrt(3.0) / 2.0))
            {
                o_data[y * MAS_DIM + x] = 3 + '0';
            }
            else
            {
                o_data[y * MAS_DIM + x] = 0 + '0';
            }
            return;
        }

        cur_re = new_re;
        cur_im = new_im;
    }

    o_data[y * MAS_DIM + x] = 0 + '0';
}

int
main(int argc, char **argv)
{
    hipError_t err = hipSuccess;

    dim3 dimsOut(MAS_DIM, MAS_DIM);
    unsigned int mem_size_Out = dimsOut.x * dimsOut.y * sizeof(unsigned char);
    unsigned char * h_Out = (unsigned char *)malloc(mem_size_Out);
    
    if (h_Out == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix!\n");
        exit(EXIT_FAILURE);
    }
    
    unsigned char * d_Out = NULL;
    err = hipMalloc((void **)&d_Out, mem_size_Out);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 threads(BLOCK_DIM, BLOCK_DIM);
    dim3 grid(dimsOut.x / threads.x, dimsOut.y / threads.y);

    printf("Computing result using CUDA Kernel...\n");

    getRoot<<<grid, threads>>>(d_Out);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch getRoot kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copying output data from the CUDA device to the host memory...\n");
    err = hipMemcpy(h_Out, d_Out, mem_size_Out, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_Out);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Writing output data on disk...\n");
    FILE * output;
    output = fopen("point_mas.txt", "w");
    for (size_t i = 0; i < MAS_DIM; ++i)
    {
        fwrite(h_Out + i * MAS_DIM, MAS_DIM, 1, output);
        fwrite("\n", 1, 1, output);
    }
    fclose(output);

    free(h_Out);

    printf("Done!\n");
    return 0;
}
